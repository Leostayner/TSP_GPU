#include "hip/hip_runtime.h"

// nvcc -std=c++11 random_sol_op2.cu -o random_sol_op2

#include <iostream>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
#include <iomanip>

#define N_SOL 10000

struct point{
    double x;
    double y;
};

__device__ double dist(point p1, point p2){
    return sqrt(pow(p1.x - p2.x, 2) +
                pow(p1.y - p2.y, 2));
}

__device__ bool isIntersecting(point p1, point p2,
                               point q1, point q2){

    return  (((q1.x-p1.x)*(p2.y-p1.y) - (q1.y-p1.y) * (p2.x-p1.x)) * 
            ((q2.x-p1.x)*(p2.y-p1.y) - (q2.y-p1.y) * (p2.x-p1.x)) < 0)
            &&
            (((p1.x-q1.x)*(q2.y-q1.y) - (p1.y-q1.y) * (q2.x-q1.x)) * 
            ((p2.x-q1.x)*(q2.y-q1.y) - (p2.y-q1.y) * (q2.x-q1.x)) < 0);
}


__global__ void path_dist(int *path, double*path_dist, double *vec_dist, point *points, int N_Sol, int N){
    int i = N * (blockIdx.x * blockDim.x + threadIdx.x);
    if(i >= N_Sol*N) return;
    
    hiprandState st;
    hiprand_init(0, i/N, 0, &st);
    
    for(int k=0; k<N; ++k)
        path[i+k] = k;  
    
    for(int k=1; k<N; ++k){
        int r = (int) ((N-k) * hiprand_uniform(&st) + k);
        
        auto tmp  = path[i+k];
        path[i+k] = path[i+r];
        path[i+r] = tmp;
        path_dist[(i/N)] += vec_dist[path[i+k-1] * N + path[i+k]];   
    }   

    path_dist[i/N] += vec_dist[path[i] * N + path[i+N-1]];
    
    //B
    bool flag = true;
    //while(flag){ 
        flag = false;
        for(int k=0; k<N-1; ++k){
            for(int j=k+1; j<N; ++j){
                int last = (j+1 == N-1)? 0 : j+1;
                
                if(isIntersecting(points[path[i+k]], points[path[i+k+1]],
                                points[path[i+j]], points[path[i+last]])){
                    
                    path_dist[int(i/N)] -= vec_dist[path[i+k] * N + path[i+k+1]] + vec_dist[path[i+k+1] * N + path[i+k+2]]+
                                        vec_dist[path[i+j-1] * N + path[i+j]] + vec_dist[path[i+j] * N + path[i+j+1]];
                    
                    auto tmp = path[i+k+1];
                    path[i+k+1]  = path[i+j];
                    path[i+j]    = tmp;
                    

                    path_dist[int(i/N)] += vec_dist[path[i+k] * N + path[i+k+1]] + vec_dist[path[i+k+1] * N + path[i+k+2]]+
                                        vec_dist[path[i+j-1] * N + path[i+j]] + vec_dist[path[i+j] * N + path[i+j+1]];
                    
                    //flag = true; 
                }
            }   
        }
    //}         
}

__global__ void points_distance(double *vec_dist, point *points, int width, int heigth){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= heigth || j >= width) return;
    
    vec_dist[i * width + j] = dist(points[i], points[j]);
}

int main(){
    std::cout << std::fixed <<std::setprecision(5);
    int N; std::cin >> N;
    thrust::host_vector<point> points_CPU;
    
    for(int i=0; i<N; ++i){
        point pt;
            std::cin >> pt.x; std::cin >> pt.y;
        points_CPU.push_back(pt);
    }

    thrust::device_vector<point> points_GPU(points_CPU);
    thrust::device_vector<double> dist_GPU(N*N, 0);
    

    dim3 dimGrid(ceil(N/32.0), ceil(N/32.0), 1);
    dim3 dimBlock(32, 32, 1);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);


    points_distance<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(dist_GPU.data()),
                                           thrust::raw_pointer_cast(points_GPU.data()),
                                           N, N);

    thrust::device_vector<int> paths(N_SOL*N, 0);
    thrust::device_vector<double> paths_dists(N_SOL, 0);

    path_dist<<<ceil(N_SOL/1024.0), 1024>>>(thrust::raw_pointer_cast(paths.data()),
                                          thrust::raw_pointer_cast(paths_dists.data()),
                                          thrust::raw_pointer_cast(dist_GPU.data()),
                                          thrust::raw_pointer_cast(points_GPU.data()), N_SOL, N);
    
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
                                      
    
    thrust::host_vector<int> paths_CPU(paths);
    thrust::host_vector<double> paths_dists_CPU(paths_dists);
                                      

    auto iter = thrust::min_element(paths_dists_CPU.begin(), paths_dists_CPU.end());
    unsigned int position = iter - paths_dists_CPU.begin();
    double max_val = *iter;

    std::cout << max_val << " 0" << std::endl;

    for (auto it = paths.begin()+(position*N); it != paths.begin()+(position*N)+N; ++it)
        std::cout << *it << " ";
 
    std::cout << std::endl;
    
    std::cerr << "Time: " << msecTotal/1000 << std::endl; 
}